#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include "Bitmap.h"

__global__ void kernel(unsigned char *inputImage, unsigned char *outputImage, int width, int height, int radius)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= height || col >= width) return;

	int hist[256];
    memset(hist, 0, sizeof(hist[0]) * 256);

	for (int i = -radius; i <= radius; ++i) { 
		for (int j = -radius; j <= radius; ++j){
            int x = row + i;
            int y = col + j;
            if (x < 0) x = -x;
            if (y < 0) y = -y;
            if (x >= height) x -= (x - height) * 2;
            if (y >= width)  y -= (y - width)  * 2;
			++hist[inputImage[x * width + y]];
		}
	}

    int bound = (2 * radius + 1) * (2 * radius + 1) / 2;
    int sum = 0;
    for (int i = 0; i < 256; ++i) {
        sum += hist[i];
        if (sum >= bound) {
            outputImage[row * width + col] = i;
            break;
        }
    }
}

void MedianFilter(Bitmap* image, Bitmap* outputImage, int radius, int blockSize) {
	int width  = image->Width();
	int height = image->Height();
	int size =  width * height * sizeof(char);

	unsigned char *deviceinputimage;
	assert(hipSuccess == hipMalloc((void**) &deviceinputimage, size));
    assert(hipSuccess == hipMemcpy(deviceinputimage, image->image, size, hipMemcpyHostToDevice));

	unsigned char *deviceOutputImage;
	hipMalloc((void**) &deviceOutputImage, size);

	dim3 dimBlock(blockSize, blockSize);
	dim3 dimGrid((width + blockSize - 1) / blockSize,
                (height + blockSize - 1) / blockSize);

	kernel<<<dimGrid, dimBlock>>>(deviceinputimage, deviceOutputImage, width, height, radius);

    assert(hipSuccess == hipMemcpy(outputImage->image, deviceOutputImage, size, hipMemcpyDeviceToHost));
	hipFree(deviceinputimage);
	hipFree(deviceOutputImage);
}

void parse_argv(int argc, char *argv[], char **inputFname, char **outputFname, int *radius, int *blockSize)
{
    static struct option long_options[] =
    {
        {"inputImage",  required_argument, NULL, 'i'},
        {"outputImage", required_argument, NULL, 'o'},
        {"radius",      optional_argument, NULL, 'r'},
        {"blockSize",   optional_argument, NULL, 'b'},
        {NULL, 0, NULL, 0}
    };

    int ch = 0;
    while ((ch = getopt_long(argc, argv, "i:o:r:b:", long_options, NULL)) != -1) {
        switch (ch) {
             case 'i' : *inputFname = optarg;
                 break;
             case 'o' : *outputFname = optarg;
                 break;
             case 'r' : *radius = atoi(optarg);
                 break;
             case 'b' : *blockSize = atoi(optarg);
                 break;
             default:
                 abort();
        }
    }
}

int main(int argc, char *argv[])
{
    char *inputFname  = NULL; 
    char *outputFname = NULL;
    int blockSize = 16;
    int radius = 1;

    parse_argv(argc, argv, &inputFname, &outputFname, &radius, &blockSize);

	Bitmap* inputImage  = new Bitmap();
	Bitmap* outputImage = new Bitmap();

	inputImage->Load(inputFname);
    outputImage->Load(inputFname);

	MedianFilter(inputImage, outputImage, radius, blockSize);
	outputImage->Save(outputFname);
}
